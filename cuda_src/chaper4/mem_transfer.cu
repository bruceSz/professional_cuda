#include "../common.h"
#include <hip/hip_runtime.h>
#include <cstdio>

int main() {
    auto prop = getCudaInfo();
    
    unsigned int size = 1<20;
    unsigned int nb = size * sizeof(float);



    float* ha, *da;
    ha = (float*)malloc(nb);

    CHECK(hipMalloc((float**)&da, nb));
    for(int i =0;i<size;i++) {
        ha[i] = 100.1f;
    }

    auto start = seconds();
    CHECK(hipMemcpy(da, ha, nb, hipMemcpyHostToDevice));
    CHECK(hipMemcpy( ha, da,nb, hipMemcpyDeviceToHost));
    auto elaps = seconds() - start;

    cout << "for array with size: " << size  << " copy from and to cost: " << elaps << endl;


    float* pha;
    CHECK(hipHostMalloc((float**)&pha, nb));

    for(int i=0;i<size;i++)  {
        pha[i] = 100.2f;
    }

    start = seconds();
    CHECK(hipMemcpy(da, pha, nb, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(pha, da, nb, hipMemcpyDeviceToHost));
    elaps = seconds() - start;

    cout << "for array with size: " << size  << " copy from and to cost:  (with pin host mem)" << elaps << endl;
    // it seems there is no much differences between above two parts of `memcpy` example
    // maybe this example is too simple

    CHECK(hipFree(da));
    free(ha);
    CHECK(hipHostFree(pha));

    hipDeviceReset();


}